#include "hip/hip_runtime.h"
﻿#ifdef WIN32
#include "hip/hip_runtime.h"
#include ""
#endif // WIN32

#include <stdlib.h>
#include <stdio.h>

#define CSC(call) 						\
do {									\
	hipError_t	status = call;			\
	if (status != hipSuccess) {		\
		fprintf(stderr, "ERROR in %s:%d. Massage: %s\n", __FILE__, __LINE__, hipGetErrorString(status));			\
		exit(0);						\
	}									\
} while(0)


__global__ void powerKernel(double* arr, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    while (idx < n) {
        arr[idx] = arr[idx] * arr[idx];
        idx += offset;
    }
}

void DefaultInput(double* arr,int n)
{
    for (int i = 0; i < n; i++)
    {
        double buff;
        scanf("%lf", &buff);
        arr[i] = buff;
    }
}

void GenerateData(double* arr, int n)
{
    for (int i = 0; i < n; i++)
    {
        arr[i] = i % 1000;
    }
}

void process(int n)
{
    double* arr = (double*)malloc(sizeof(double) * n);
    // DefaultInput(arr, n);
    GenerateData(arr, n);

    double* cudaArr;
    CSC(hipMalloc(&cudaArr, sizeof(double) * n));
    CSC(hipMemcpy(cudaArr, arr, sizeof(double) * n, hipMemcpyHostToDevice));

    hipEvent_t start, end;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&end));
    CSC(hipEventRecord(start));

    powerKernel << <1024, 1024 >> > (cudaArr, n);
    CSC(hipGetLastError());

    CSC(hipEventRecord(end));
    CSC(hipEventSynchronize(end));
    float t;
    CSC(hipEventElapsedTime(&t, start, end));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(end));

    printf("time = %f ms\n", t);

    CSC(hipMemcpy(arr, cudaArr, sizeof(double) * n, hipMemcpyDeviceToHost));
    CSC(hipFree(cudaArr));

    /*
    for (int i = 0; i < n; i++)
        printf("%.10e ", arr[i]);
    printf("\n");
    */
    free(arr);
}

int main()
{
    process(1000);
    process(10000);
    process(100000);
    process(1000000);
    process(10000000);
    process(100000000);
    return 0;
}
